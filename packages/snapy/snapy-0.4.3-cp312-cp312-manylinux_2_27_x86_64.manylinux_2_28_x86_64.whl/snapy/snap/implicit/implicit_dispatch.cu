// eigen
#include <Eigen/Dense>

// torch
#include <ATen/Dispatch.h>
#include <ATen/TensorIterator.h>
#include <ATen/native/ReduceOpsUtils.h>
#include <c10/cuda/CUDAGuard.h>
#include <c10/core/ScalarType.h>

// snap
#include <snap/utils/cuda_utils.h>
#include <snap/loops.cuh>
#include "tridiag_thomas_impl.h"
#include "implicit_dispatch.hpp"

namespace snap {

template <int N>
void vic_solve_cuda(at::TensorIterator& iter, double dt, int il, int iu) {
  at::cuda::CUDAGuard device_guard(iter.device());

  AT_DISPATCH_FLOATING_TYPES(iter.dtype(), "vic_solve_cuda", [&]() {
    auto nhydro = at::native::ensure_nonempty_size(iter.output(), 0);
    auto stride = at::native::ensure_nonempty_stride(iter.output(), 0);

    native::gpu_kernel<7>(iter, [=] GPU_LAMBDA(
                                              char* const data[7],
                                              unsigned int strides[7]) {
      auto du = reinterpret_cast<scalar_t*>(data[0] + strides[0]);
      auto w = reinterpret_cast<scalar_t*>(data[1] + strides[1]);
      auto a =
          reinterpret_cast<Eigen::Matrix<scalar_t, N, N, Eigen::RowMajor>*>(
              data[2] + strides[2]);
      auto b =
          reinterpret_cast<Eigen::Matrix<scalar_t, N, N, Eigen::RowMajor>*>(
              data[3] + strides[3]);
      auto c =
          reinterpret_cast<Eigen::Matrix<scalar_t, N, N, Eigen::RowMajor>*>(
              data[4] + strides[4]);
      auto delta =
          reinterpret_cast<Eigen::Vector<scalar_t, N>*>(data[5] + strides[5]);
      auto corr =
          reinterpret_cast<Eigen::Vector<scalar_t, N>*>(data[6] + strides[6]);

      forward_sweep_impl(a, b, c, delta, corr, du, dt, nhydro, stride, il, iu);
      backward_substitution_impl(a, delta, w, du, nhydro, stride, il, iu);
    });
  });
}

template <int N>
void alloc_eigen_cuda(c10::ScalarType dtype,
                      char *&a, char *&b, char *&c, char *&delta, char *&corr,
                      int ncol, int nlayer) {
  AT_DISPATCH_FLOATING_TYPES(dtype, "alloc_eigen_cuda", [&]() {
    hipMalloc(
        (void **)&a,
        sizeof(Eigen::Matrix<scalar_t, N, N, Eigen::RowMajor>) * ncol * nlayer);
    int err = checkCudaError("alloc_eigen_cuda::a");
    TORCH_CHECK(err == 0, "eigen memory allocation error");

    hipMalloc(
        (void **)&b,
        sizeof(Eigen::Matrix<scalar_t, N, N, Eigen::RowMajor>) * ncol * nlayer);
    err = checkCudaError("alloc_eigen_cuda::b");
    TORCH_CHECK(err == 0, "eigen memory allocation error");

    hipMalloc(
        (void **)&c,
        sizeof(Eigen::Matrix<scalar_t, N, N, Eigen::RowMajor>) * ncol * nlayer);
    err = checkCudaError("alloc_eigen_cuda::c");
    TORCH_CHECK(err == 0, "eigen memory allocation error");

    hipMalloc((void **)&delta,
               sizeof(Eigen::Vector<scalar_t, N>) * ncol * nlayer);
    err = checkCudaError("alloc_eigen_cuda::delta");
    TORCH_CHECK(err == 0, "eigen memory allocation error");

    hipMalloc((void **)&corr,
               sizeof(Eigen::Vector<scalar_t, N>) * ncol * nlayer);
    err = checkCudaError("alloc_eigen_cuda::corr");
    TORCH_CHECK(err == 0, "eigen memory allocation error");
  });
}

void free_eigen_cuda(char *&a, char *&b, char *&c, char *&delta, char *&corr) {
  hipDeviceSynchronize();
  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipFree(delta);
  hipFree(corr);
}

}  // namespace snap

namespace at::native {

REGISTER_CUDA_DISPATCH(vic_solve3, &snap::vic_solve_cuda<3>);
REGISTER_CUDA_DISPATCH(vic_solve5, &snap::vic_solve_cuda<5>);

REGISTER_CUDA_DISPATCH(alloc_eigen3, &snap::alloc_eigen_cuda<3>);
REGISTER_CUDA_DISPATCH(alloc_eigen5, &snap::alloc_eigen_cuda<5>);
REGISTER_CUDA_DISPATCH(free_eigen, &snap::free_eigen_cuda);

}  // namespace at::native
